#include "hip/hip_runtime.h"
/*M///////////////////////////////////////////////////////////////////////////////////////
//
//  IMPORTANT: READ BEFORE DOWNLOADING, COPYING, INSTALLING OR USING.
//
//  By downloading, copying, installing or using the software you agree to this license.
//  If you do not agree to this license, do not download, install,
//  copy or use the software.
//
//
//                           License Agreement
//                For Open Source Computer Vision Library
//
// Copyright (C) 2000-2008, Intel Corporation, all rights reserved.
// Copyright (C) 2009, Willow Garage Inc., all rights reserved.
// Third party copyrights are property of their respective owners.
//
// Redistribution and use in source and binary forms, with or without modification,
// are permitted provided that the following conditions are met:
//
//   * Redistribution's of source code must retain the above copyright notice,
//     this list of conditions and the following disclaimer.
//
//   * Redistribution's in binary form must reproduce the above copyright notice,
//     this list of conditions and the following disclaimer in the documentation
//     and/or other materials provided with the distribution.
//
//   * The name of the copyright holders may not be used to endorse or promote products
//     derived from this software without specific prior written permission.
//
// This software is provided by the copyright holders and contributors "as is" and
// any express or implied warranties, including, but not limited to, the implied
// warranties of merchantability and fitness for a particular purpose are disclaimed.
// In no event shall the Intel Corporation or contributors be liable for any direct,
// indirect, incidental, special, exemplary, or consequential damages
// (including, but not limited to, procurement of substitute goods or services;
// loss of use, data, or profits; or business interruption) however caused
// and on any theory of liability, whether in contract, strict liability,
// or tort (including negligence or otherwise) arising in any way out of
// the use of this software, even if advised of the possibility of such damage.
//
//M*/

#if !defined CUDA_DISABLER

#include <thrust/device_ptr.h>
#include <thrust/sort.h>

#include "opencv2/gpu/device/common.hpp"
#include "opencv2/gpu/device/emulation.hpp"
#include "opencv2/gpu/device/vec_math.hpp"
#include "opencv2/gpu/device/functional.hpp"
#include "opencv2/gpu/device/limits.hpp"
#include "opencv2/gpu/device/dynamic_smem.hpp"

namespace cv { namespace gpu { namespace device
{
    namespace hough
    {
        __device__ int g_counter;

        ////////////////////////////////////////////////////////////////////////
        // buildPointList

        template <int PIXELS_PER_THREAD>
        __global__ void buildPointList(const PtrStepSzb src, unsigned int* list)
        {
            __shared__ unsigned int s_queues[4][32 * PIXELS_PER_THREAD];
            __shared__ int s_qsize[4];
            __shared__ int s_globStart[4];

            const int x = blockIdx.x * blockDim.x * PIXELS_PER_THREAD + threadIdx.x;
            const int y = blockIdx.y * blockDim.y + threadIdx.y;

            if (threadIdx.x == 0)
                s_qsize[threadIdx.y] = 0;
            __syncthreads();

            if (y < src.rows)
            {
                // fill the queue
                const uchar* srcRow = src.ptr(y);
                for (int i = 0, xx = x; i < PIXELS_PER_THREAD && xx < src.cols; ++i, xx += blockDim.x)
                {
                    if (srcRow[xx])
                    {
                        const unsigned int val = (y << 16) | xx;
                        const int qidx = Emulation::smem::atomicAdd(&s_qsize[threadIdx.y], 1);
                        s_queues[threadIdx.y][qidx] = val;
                    }
                }
            }

            __syncthreads();

            // let one thread reserve the space required in the global list
            if (threadIdx.x == 0 && threadIdx.y == 0)
            {
                // find how many items are stored in each list
                int totalSize = 0;
                for (int i = 0; i < blockDim.y; ++i)
                {
                    s_globStart[i] = totalSize;
                    totalSize += s_qsize[i];
                }

                // calculate the offset in the global list
                const int globalOffset = atomicAdd(&g_counter, totalSize);
                for (int i = 0; i < blockDim.y; ++i)
                    s_globStart[i] += globalOffset;
            }

            __syncthreads();

            // copy local queues to global queue
            const int qsize = s_qsize[threadIdx.y];
            int gidx = s_globStart[threadIdx.y] + threadIdx.x;
            for(int i = threadIdx.x; i < qsize; i += blockDim.x, gidx += blockDim.x)
                list[gidx] = s_queues[threadIdx.y][i];
        }

        int buildPointList_gpu(PtrStepSzb src, unsigned int* list)
        {
            const int PIXELS_PER_THREAD = 16;

            void* counterPtr;
            cudaSafeCall( hipGetSymbolAddress(&counterPtr, HIP_SYMBOL(g_counter)) );

            cudaSafeCall( hipMemset(counterPtr, 0, sizeof(int)) );

            const dim3 block(32, 4);
            const dim3 grid(divUp(src.cols, block.x * PIXELS_PER_THREAD), divUp(src.rows, block.y));

            cudaSafeCall( hipFuncSetCacheConfig(reinterpret_cast<const void*>(buildPointList<PIXELS_PER_THREAD>), hipFuncCachePreferShared) );

            buildPointList<PIXELS_PER_THREAD><<<grid, block>>>(src, list);
            cudaSafeCall( hipGetLastError() );

            cudaSafeCall( hipDeviceSynchronize() );

            int totalCount;
            cudaSafeCall( hipMemcpy(&totalCount, counterPtr, sizeof(int), hipMemcpyDeviceToHost) );

            return totalCount;
        }

        ////////////////////////////////////////////////////////////////////////
        // linesAccum

        __global__ void linesAccumGlobal(const unsigned int* list, const int count, PtrStepi accum, const float irho, const float theta, const int numrho)
        {
            const int n = blockIdx.x;
            const float ang = n * theta;

            float sinVal;
            float cosVal;
            sincosf(ang, &sinVal, &cosVal);
            sinVal *= irho;
            cosVal *= irho;

            const int shift = (numrho - 1) / 2;

            int* accumRow = accum.ptr(n + 1);
            for (int i = threadIdx.x; i < count; i += blockDim.x)
            {
                const unsigned int val = list[i];

                const int x = (val & 0xFFFF);
                const int y = (val >> 16) & 0xFFFF;

                int r = __float2int_rn(x * cosVal + y * sinVal);
                r += shift;

                ::atomicAdd(accumRow + r + 1, 1);
            }
        }

        __global__ void linesAccumShared(const unsigned int* list, const int count, PtrStepi accum, const float irho, const float theta, const int numrho)
        {
            int* smem = DynamicSharedMem<int>();

            for (int i = threadIdx.x; i < numrho + 1; i += blockDim.x)
                smem[i] = 0;

            __syncthreads();

            const int n = blockIdx.x;
            const float ang = n * theta;

            float sinVal;
            float cosVal;
            sincosf(ang, &sinVal, &cosVal);
            sinVal *= irho;
            cosVal *= irho;

            const int shift = (numrho - 1) / 2;

            for (int i = threadIdx.x; i < count; i += blockDim.x)
            {
                const unsigned int val = list[i];

                const int x = (val & 0xFFFF);
                const int y = (val >> 16) & 0xFFFF;

                int r = __float2int_rn(x * cosVal + y * sinVal);
                r += shift;

                Emulation::smem::atomicAdd(&smem[r + 1], 1);
            }

            __syncthreads();

            int* accumRow = accum.ptr(n + 1);
            for (int i = threadIdx.x; i < numrho + 1; i += blockDim.x)
                accumRow[i] = smem[i];
        }

        void linesAccum_gpu(const unsigned int* list, int count, PtrStepSzi accum, float rho, float theta, size_t sharedMemPerBlock, bool has20)
        {
            const dim3 block(has20 ? 1024 : 512);
            const dim3 grid(accum.rows - 2);

            size_t smemSize = (accum.cols - 1) * sizeof(int);

            if (smemSize < sharedMemPerBlock - 1000)
                linesAccumShared<<<grid, block, smemSize>>>(list, count, accum, 1.0f / rho, theta, accum.cols - 2);
            else
                linesAccumGlobal<<<grid, block>>>(list, count, accum, 1.0f / rho, theta, accum.cols - 2);

            cudaSafeCall( hipGetLastError() );

            cudaSafeCall( hipDeviceSynchronize() );
        }

        ////////////////////////////////////////////////////////////////////////
        // linesGetResult

        __global__ void linesGetResult(const PtrStepSzi accum, float2* out, int* votes, const int maxSize, const float rho, const float theta, const int threshold, const int numrho)
        {
            const int r = blockIdx.x * blockDim.x + threadIdx.x;
            const int n = blockIdx.y * blockDim.y + threadIdx.y;

            if (r >= accum.cols - 2 || n >= accum.rows - 2)
                return;

            const int curVotes = accum(n + 1, r + 1);

            if (curVotes > threshold &&
                curVotes >  accum(n + 1, r) &&
                curVotes >= accum(n + 1, r + 2) &&
                curVotes >  accum(n, r + 1) &&
                curVotes >= accum(n + 2, r + 1))
            {
                const float radius = (r - (numrho - 1) * 0.5f) * rho;
                const float angle = n * theta;

                const int ind = ::atomicAdd(&g_counter, 1);
                if (ind < maxSize)
                {
                    out[ind] = make_float2(radius, angle);
                    votes[ind] = curVotes;
                }
            }
        }

        int linesGetResult_gpu(PtrStepSzi accum, float2* out, int* votes, int maxSize, float rho, float theta, int threshold, bool doSort)
        {
            void* counterPtr;
            cudaSafeCall( hipGetSymbolAddress(&counterPtr, HIP_SYMBOL(g_counter)) );

            cudaSafeCall( hipMemset(counterPtr, 0, sizeof(int)) );

            const dim3 block(32, 8);
            const dim3 grid(divUp(accum.cols - 2, block.x), divUp(accum.rows - 2, block.y));

            cudaSafeCall( hipFuncSetCacheConfig(reinterpret_cast<const void*>(linesGetResult), hipFuncCachePreferL1) );

            linesGetResult<<<grid, block>>>(accum, out, votes, maxSize, rho, theta, threshold, accum.cols - 2);
            cudaSafeCall( hipGetLastError() );

            cudaSafeCall( hipDeviceSynchronize() );

            int totalCount;
            cudaSafeCall( hipMemcpy(&totalCount, counterPtr, sizeof(int), hipMemcpyDeviceToHost) );

            totalCount = ::min(totalCount, maxSize);

            if (doSort && totalCount > 0)
            {
                thrust::device_ptr<float2> outPtr(out);
                thrust::device_ptr<int> votesPtr(votes);
                thrust::sort_by_key(votesPtr, votesPtr + totalCount, outPtr, thrust::greater<int>());
            }

            return totalCount;
        }

        ////////////////////////////////////////////////////////////////////////
        // houghLinesProbabilistic

        texture<uchar, hipTextureType2D, hipReadModeElementType> tex_mask(false, hipFilterModePoint, hipAddressModeClamp);

        __global__ void houghLinesProbabilistic(const PtrStepSzi accum,
                                                int4* out, const int maxSize,
                                                const float rho, const float theta,
                                                const int lineGap, const int lineLength,
                                                const int rows, const int cols)
        {
            const int r = blockIdx.x * blockDim.x + threadIdx.x;
            const int n = blockIdx.y * blockDim.y + threadIdx.y;

            if (r >= accum.cols - 2 || n >= accum.rows - 2)
                return;

            const int curVotes = accum(n + 1, r + 1);

            if (curVotes >= lineLength &&
                curVotes > accum(n, r) &&
                curVotes > accum(n, r + 1) &&
                curVotes > accum(n, r + 2) &&
                curVotes > accum(n + 1, r) &&
                curVotes > accum(n + 1, r + 2) &&
                curVotes > accum(n + 2, r) &&
                curVotes > accum(n + 2, r + 1) &&
                curVotes > accum(n + 2, r + 2))
            {
                const float radius = (r - (accum.cols - 2 - 1) * 0.5f) * rho;
                const float angle = n * theta;

                float cosa;
                float sina;
                sincosf(angle, &sina, &cosa);

                float2 p0 = make_float2(cosa * radius, sina * radius);
                float2 dir = make_float2(-sina, cosa);

                float2 pb[4] = {make_float2(-1, -1), make_float2(-1, -1), make_float2(-1, -1), make_float2(-1, -1)};
                float a;

                if (dir.x != 0)
                {
                    a = -p0.x / dir.x;
                    pb[0].x = 0;
                    pb[0].y = p0.y + a * dir.y;

                    a = (cols - 1 - p0.x) / dir.x;
                    pb[1].x = cols - 1;
                    pb[1].y = p0.y + a * dir.y;
                }
                if (dir.y != 0)
                {
                    a = -p0.y / dir.y;
                    pb[2].x = p0.x + a * dir.x;
                    pb[2].y = 0;

                    a = (rows - 1 - p0.y) / dir.y;
                    pb[3].x = p0.x + a * dir.x;
                    pb[3].y = rows - 1;
                }

                if (pb[0].x == 0 && (pb[0].y >= 0 && pb[0].y < rows))
                {
                    p0 = pb[0];
                    if (dir.x < 0)
                        dir = -dir;
                }
                else if (pb[1].x == cols - 1 && (pb[0].y >= 0 && pb[0].y < rows))
                {
                    p0 = pb[1];
                    if (dir.x > 0)
                        dir = -dir;
                }
                else if (pb[2].y == 0 && (pb[2].x >= 0 && pb[2].x < cols))
                {
                    p0 = pb[2];
                    if (dir.y < 0)
                        dir = -dir;
                }
                else if (pb[3].y == rows - 1 && (pb[3].x >= 0 && pb[3].x < cols))
                {
                    p0 = pb[3];
                    if (dir.y > 0)
                        dir = -dir;
                }

                float2 d;
                if (::fabsf(dir.x) > ::fabsf(dir.y))
                {
                    d.x = dir.x > 0 ? 1 : -1;
                    d.y = dir.y / ::fabsf(dir.x);
                }
                else
                {
                    d.x = dir.x / ::fabsf(dir.y);
                    d.y = dir.y > 0 ? 1 : -1;
                }

                float2 line_end[2];
                int gap;
                bool inLine = false;

                float2 p1 = p0;
                if (p1.x < 0 || p1.x >= cols || p1.y < 0 || p1.y >= rows)
                    return;

                for (;;)
                {
                    if (tex2D(tex_mask, p1.x, p1.y))
                    {
                        gap = 0;

                        if (!inLine)
                        {
                            line_end[0] = p1;
                            line_end[1] = p1;
                            inLine = true;
                        }
                        else
                        {
                            line_end[1] = p1;
                        }
                    }
                    else if (inLine)
                    {
                        if (++gap > lineGap)
                        {
                            bool good_line = ::abs(line_end[1].x - line_end[0].x) >= lineLength ||
                                             ::abs(line_end[1].y - line_end[0].y) >= lineLength;

                            if (good_line)
                            {
                                const int ind = ::atomicAdd(&g_counter, 1);
                                if (ind < maxSize)
                                    out[ind] = make_int4(line_end[0].x, line_end[0].y, line_end[1].x, line_end[1].y);
                            }

                            gap = 0;
                            inLine = false;
                        }
                    }

                    p1 = p1 + d;
                    if (p1.x < 0 || p1.x >= cols || p1.y < 0 || p1.y >= rows)
                    {
                        if (inLine)
                        {
                            bool good_line = ::abs(line_end[1].x - line_end[0].x) >= lineLength ||
                                             ::abs(line_end[1].y - line_end[0].y) >= lineLength;

                            if (good_line)
                            {
                                const int ind = ::atomicAdd(&g_counter, 1);
                                if (ind < maxSize)
                                    out[ind] = make_int4(line_end[0].x, line_end[0].y, line_end[1].x, line_end[1].y);
                            }

                        }
                        break;
                    }
                }
            }
        }

        int houghLinesProbabilistic_gpu(PtrStepSzb mask, PtrStepSzi accum, int4* out, int maxSize, float rho, float theta, int lineGap, int lineLength)
        {
            void* counterPtr;
            cudaSafeCall( hipGetSymbolAddress(&counterPtr, HIP_SYMBOL(g_counter)) );

            cudaSafeCall( hipMemset(counterPtr, 0, sizeof(int)) );

            const dim3 block(32, 8);
            const dim3 grid(divUp(accum.cols - 2, block.x), divUp(accum.rows - 2, block.y));

            bindTexture(&tex_mask, mask);

            houghLinesProbabilistic<<<grid, block>>>(accum,
                                                     out, maxSize,
                                                     rho, theta,
                                                     lineGap, lineLength,
                                                     mask.rows, mask.cols);
            cudaSafeCall( hipGetLastError() );

            cudaSafeCall( hipDeviceSynchronize() );

            int totalCount;
            cudaSafeCall( hipMemcpy(&totalCount, counterPtr, sizeof(int), hipMemcpyDeviceToHost) );

            totalCount = ::min(totalCount, maxSize);

            return totalCount;
        }

        ////////////////////////////////////////////////////////////////////////
        // circlesAccumCenters

        __global__ void circlesAccumCenters(const unsigned int* list, const int count, const PtrStepi dx, const PtrStepi dy,
                                            PtrStepi accum, const int width, const int height, const int minRadius, const int maxRadius, const float idp)
        {
            const int SHIFT = 10;
            const int ONE = 1 << SHIFT;

            const int tid = blockIdx.x * blockDim.x + threadIdx.x;

            if (tid >= count)
                return;

            const unsigned int val = list[tid];

            const int x = (val & 0xFFFF);
            const int y = (val >> 16) & 0xFFFF;

            const int vx = dx(y, x);
            const int vy = dy(y, x);

            if (vx == 0 && vy == 0)
                return;

            const float mag = ::sqrtf(vx * vx + vy * vy);

            const int x0 = __float2int_rn((x * idp) * ONE);
            const int y0 = __float2int_rn((y * idp) * ONE);

            int sx = __float2int_rn((vx * idp) * ONE / mag);
            int sy = __float2int_rn((vy * idp) * ONE / mag);

            // Step from minRadius to maxRadius in both directions of the gradient
            for (int k1 = 0; k1 < 2; ++k1)
            {
                int x1 = x0 + minRadius * sx;
                int y1 = y0 + minRadius * sy;

                for (int r = minRadius; r <= maxRadius; x1 += sx, y1 += sy, ++r)
                {
                    const int x2 = x1 >> SHIFT;
                    const int y2 = y1 >> SHIFT;

                    if (x2 < 0 || x2 >= width || y2 < 0 || y2 >= height)
                        break;

                    ::atomicAdd(accum.ptr(y2 + 1) + x2 + 1, 1);
                }

                sx = -sx;
                sy = -sy;
            }
        }

        void circlesAccumCenters_gpu(const unsigned int* list, int count, PtrStepi dx, PtrStepi dy, PtrStepSzi accum, int minRadius, int maxRadius, float idp)
        {
            const dim3 block(256);
            const dim3 grid(divUp(count, block.x));

            cudaSafeCall( hipFuncSetCacheConfig(reinterpret_cast<const void*>(circlesAccumCenters), hipFuncCachePreferL1) );

            circlesAccumCenters<<<grid, block>>>(list, count, dx, dy, accum, accum.cols - 2, accum.rows - 2, minRadius, maxRadius, idp);
            cudaSafeCall( hipGetLastError() );

            cudaSafeCall( hipDeviceSynchronize() );
        }

        ////////////////////////////////////////////////////////////////////////
        // buildCentersList

        __global__ void buildCentersList(const PtrStepSzi accum, unsigned int* centers, const int threshold)
        {
            const int x = blockIdx.x * blockDim.x + threadIdx.x;
            const int y = blockIdx.y * blockDim.y + threadIdx.y;

            if (x < accum.cols - 2 && y < accum.rows - 2)
            {
                const int top = accum(y, x + 1);

                const int left = accum(y + 1, x);
                const int cur = accum(y + 1, x + 1);
                const int right = accum(y + 1, x + 2);

                const int bottom = accum(y + 2, x + 1);

                if (cur > threshold && cur > top && cur >= bottom && cur >  left && cur >= right)
                {
                    const unsigned int val = (y << 16) | x;
                    const int idx = ::atomicAdd(&g_counter, 1);
                    centers[idx] = val;
                }
            }
        }

        int buildCentersList_gpu(PtrStepSzi accum, unsigned int* centers, int threshold)
        {
            void* counterPtr;
            cudaSafeCall( hipGetSymbolAddress(&counterPtr, HIP_SYMBOL(g_counter)) );

            cudaSafeCall( hipMemset(counterPtr, 0, sizeof(int)) );

            const dim3 block(32, 8);
            const dim3 grid(divUp(accum.cols - 2, block.x), divUp(accum.rows - 2, block.y));

            cudaSafeCall( hipFuncSetCacheConfig(reinterpret_cast<const void*>(buildCentersList), hipFuncCachePreferL1) );

            buildCentersList<<<grid, block>>>(accum, centers, threshold);
            cudaSafeCall( hipGetLastError() );

            cudaSafeCall( hipDeviceSynchronize() );

            int totalCount;
            cudaSafeCall( hipMemcpy(&totalCount, counterPtr, sizeof(int), hipMemcpyDeviceToHost) );

            return totalCount;
        }

        ////////////////////////////////////////////////////////////////////////
        // circlesAccumRadius

        __global__ void circlesAccumRadius(const unsigned int* centers, const unsigned int* list, const int count,
                                           float3* circles, const int maxCircles, const float dp,
                                           const int minRadius, const int maxRadius, const int histSize, const int threshold)
        {
            int* smem = DynamicSharedMem<int>();

            for (int i = threadIdx.x; i < histSize + 2; i += blockDim.x)
                smem[i] = 0;
            __syncthreads();

            unsigned int val = centers[blockIdx.x];

            float cx = (val & 0xFFFF);
            float cy = (val >> 16) & 0xFFFF;

            cx = (cx + 0.5f) * dp;
            cy = (cy + 0.5f) * dp;

            for (int i = threadIdx.x; i < count; i += blockDim.x)
            {
                val = list[i];

                const int x = (val & 0xFFFF);
                const int y = (val >> 16) & 0xFFFF;

                const float rad = ::sqrtf((cx - x) * (cx - x) + (cy - y) * (cy - y));
                if (rad >= minRadius && rad <= maxRadius)
                {
                    const int r = __float2int_rn(rad - minRadius);

                    Emulation::smem::atomicAdd(&smem[r + 1], 1);
                }
            }

            __syncthreads();

            for (int i = threadIdx.x; i < histSize; i += blockDim.x)
            {
                const int curVotes = smem[i + 1];

                if (curVotes >= threshold && curVotes > smem[i] && curVotes >= smem[i + 2])
                {
                    const int ind = ::atomicAdd(&g_counter, 1);
                    if (ind < maxCircles)
                        circles[ind] = make_float3(cx, cy, i + minRadius);
                }
            }
        }

        int circlesAccumRadius_gpu(const unsigned int* centers, int centersCount, const unsigned int* list, int count,
                                   float3* circles, int maxCircles, float dp, int minRadius, int maxRadius, int threshold, bool has20)
        {
            void* counterPtr;
            cudaSafeCall( hipGetSymbolAddress(&counterPtr, HIP_SYMBOL(g_counter)) );

            cudaSafeCall( hipMemset(counterPtr, 0, sizeof(int)) );

            const dim3 block(has20 ? 1024 : 512);
            const dim3 grid(centersCount);

            const int histSize = maxRadius - minRadius + 1;
            size_t smemSize = (histSize + 2) * sizeof(int);

            circlesAccumRadius<<<grid, block, smemSize>>>(centers, list, count, circles, maxCircles, dp, minRadius, maxRadius, histSize, threshold);
            cudaSafeCall( hipGetLastError() );

            cudaSafeCall( hipDeviceSynchronize() );

            int totalCount;
            cudaSafeCall( hipMemcpy(&totalCount, counterPtr, sizeof(int), hipMemcpyDeviceToHost) );

            totalCount = ::min(totalCount, maxCircles);

            return totalCount;
        }

        ////////////////////////////////////////////////////////////////////////
        // Generalized Hough

        template <typename T, int PIXELS_PER_THREAD>
        __global__ void buildEdgePointList(const PtrStepSzb edges, const PtrStep<T> dx, const PtrStep<T> dy, unsigned int* coordList, float* thetaList)
        {
            __shared__ unsigned int s_coordLists[4][32 * PIXELS_PER_THREAD];
            __shared__ float s_thetaLists[4][32 * PIXELS_PER_THREAD];
            __shared__ int s_sizes[4];
            __shared__ int s_globStart[4];

            const int x = blockIdx.x * blockDim.x * PIXELS_PER_THREAD + threadIdx.x;
            const int y = blockIdx.y * blockDim.y + threadIdx.y;

            if (threadIdx.x == 0)
                s_sizes[threadIdx.y] = 0;
            __syncthreads();

            if (y < edges.rows)
            {
                // fill the queue
                const uchar* edgesRow = edges.ptr(y);
                const T* dxRow = dx.ptr(y);
                const T* dyRow = dy.ptr(y);

                for (int i = 0, xx = x; i < PIXELS_PER_THREAD && xx < edges.cols; ++i, xx += blockDim.x)
                {
                    const T dxVal = dxRow[xx];
                    const T dyVal = dyRow[xx];

                    if (edgesRow[xx] && (dxVal != 0 || dyVal != 0))
                    {
                        const unsigned int coord = (y << 16) | xx;

                        float theta = ::atan2f(dyVal, dxVal);
                        if (theta < 0)
                            theta += 2.0f * CV_PI_F;

                        const int qidx = Emulation::smem::atomicAdd(&s_sizes[threadIdx.y], 1);

                        s_coordLists[threadIdx.y][qidx] = coord;
                        s_thetaLists[threadIdx.y][qidx] = theta;
                    }
                }
            }

            __syncthreads();

            // let one thread reserve the space required in the global list
            if (threadIdx.x == 0 && threadIdx.y == 0)
            {
                // find how many items are stored in each list
                int totalSize = 0;
                for (int i = 0; i < blockDim.y; ++i)
                {
                    s_globStart[i] = totalSize;
                    totalSize += s_sizes[i];
                }

                // calculate the offset in the global list
                const int globalOffset = atomicAdd(&g_counter, totalSize);
                for (int i = 0; i < blockDim.y; ++i)
                    s_globStart[i] += globalOffset;
            }

            __syncthreads();

            // copy local queues to global queue
            const int qsize = s_sizes[threadIdx.y];
            int gidx = s_globStart[threadIdx.y] + threadIdx.x;
            for(int i = threadIdx.x; i < qsize; i += blockDim.x, gidx += blockDim.x)
            {
                coordList[gidx] = s_coordLists[threadIdx.y][i];
                thetaList[gidx] = s_thetaLists[threadIdx.y][i];
            }
        }

        template <typename T>
        int buildEdgePointList_gpu(PtrStepSzb edges, PtrStepSzb dx, PtrStepSzb dy, unsigned int* coordList, float* thetaList)
        {
            const int PIXELS_PER_THREAD = 8;

            void* counterPtr;
            cudaSafeCall( hipGetSymbolAddress(&counterPtr, HIP_SYMBOL(g_counter)) );

            cudaSafeCall( hipMemset(counterPtr, 0, sizeof(int)) );

            const dim3 block(32, 4);
            const dim3 grid(divUp(edges.cols, block.x * PIXELS_PER_THREAD), divUp(edges.rows, block.y));

            cudaSafeCall( hipFuncSetCacheConfig(reinterpret_cast<const void*>(buildEdgePointList<T), PIXELS_PER_THREAD>, hipFuncCachePreferShared) );

            buildEdgePointList<T, PIXELS_PER_THREAD><<<grid, block>>>(edges, (PtrStepSz<T>) dx, (PtrStepSz<T>) dy, coordList, thetaList);
            cudaSafeCall( hipGetLastError() );

            cudaSafeCall( hipDeviceSynchronize() );

            int totalCount;
            cudaSafeCall( hipMemcpy(&totalCount, counterPtr, sizeof(int), hipMemcpyDeviceToHost) );

            return totalCount;
        }

        template int buildEdgePointList_gpu<short>(PtrStepSzb edges, PtrStepSzb dx, PtrStepSzb dy, unsigned int* coordList, float* thetaList);
        template int buildEdgePointList_gpu<int>(PtrStepSzb edges, PtrStepSzb dx, PtrStepSzb dy, unsigned int* coordList, float* thetaList);
        template int buildEdgePointList_gpu<float>(PtrStepSzb edges, PtrStepSzb dx, PtrStepSzb dy, unsigned int* coordList, float* thetaList);

        __global__ void buildRTable(const unsigned int* coordList, const float* thetaList, const int pointsCount,
                                    PtrStep<short2> r_table, int* r_sizes, int maxSize,
                                    const short2 templCenter, const float thetaScale)
        {
            const int tid = blockIdx.x * blockDim.x + threadIdx.x;

            if (tid >= pointsCount)
                return;

            const unsigned int coord = coordList[tid];
            short2 p;
            p.x = (coord & 0xFFFF);
            p.y = (coord >> 16) & 0xFFFF;

            const float theta = thetaList[tid];
            const int n = __float2int_rn(theta * thetaScale);

            const int ind = ::atomicAdd(r_sizes + n, 1);
            if (ind < maxSize)
                r_table(n, ind) = saturate_cast<short2>(p - templCenter);
        }

        void buildRTable_gpu(const unsigned int* coordList, const float* thetaList, int pointsCount,
                             PtrStepSz<short2> r_table, int* r_sizes,
                             short2 templCenter, int levels)
        {
            const dim3 block(256);
            const dim3 grid(divUp(pointsCount, block.x));

            const float thetaScale = levels / (2.0f * CV_PI_F);

            buildRTable<<<grid, block>>>(coordList, thetaList, pointsCount, r_table, r_sizes, r_table.cols, templCenter, thetaScale);
            cudaSafeCall( hipGetLastError() );

            cudaSafeCall( hipDeviceSynchronize() );
        }

        ////////////////////////////////////////////////////////////////////////
        // GHT_Ballard_Pos

        __global__ void GHT_Ballard_Pos_calcHist(const unsigned int* coordList, const float* thetaList, const int pointsCount,
                                                 const PtrStep<short2> r_table, const int* r_sizes,
                                                 PtrStepSzi hist,
                                                 const float idp, const float thetaScale)
        {
            const int tid = blockIdx.x * blockDim.x + threadIdx.x;

            if (tid >= pointsCount)
                return;

            const unsigned int coord = coordList[tid];
            short2 p;
            p.x = (coord & 0xFFFF);
            p.y = (coord >> 16) & 0xFFFF;

            const float theta = thetaList[tid];
            const int n = __float2int_rn(theta * thetaScale);

            const short2* r_row = r_table.ptr(n);
            const int r_row_size = r_sizes[n];

            for (int j = 0; j < r_row_size; ++j)
            {
                int2 c = p - r_row[j];

                c.x = __float2int_rn(c.x * idp);
                c.y = __float2int_rn(c.y * idp);

                if (c.x >= 0 && c.x < hist.cols - 2 && c.y >= 0 && c.y < hist.rows - 2)
                    ::atomicAdd(hist.ptr(c.y + 1) + c.x + 1, 1);
            }
        }

        void GHT_Ballard_Pos_calcHist_gpu(const unsigned int* coordList, const float* thetaList, int pointsCount,
                                          PtrStepSz<short2> r_table, const int* r_sizes,
                                          PtrStepSzi hist,
                                          float dp, int levels)
        {
            const dim3 block(256);
            const dim3 grid(divUp(pointsCount, block.x));

            const float idp = 1.0f / dp;
            const float thetaScale = levels / (2.0f * CV_PI_F);

            GHT_Ballard_Pos_calcHist<<<grid, block>>>(coordList, thetaList, pointsCount, r_table, r_sizes, hist, idp, thetaScale);
            cudaSafeCall( hipGetLastError() );

            cudaSafeCall( hipDeviceSynchronize() );
        }

        __global__ void GHT_Ballard_Pos_findPosInHist(const PtrStepSzi hist, float4* out, int3* votes, const int maxSize, const float dp, const int threshold)
        {
            const int x = blockIdx.x * blockDim.x + threadIdx.x;
            const int y = blockIdx.y * blockDim.y + threadIdx.y;

            if (x >= hist.cols - 2 || y >= hist.rows - 2)
                return;

            const int curVotes = hist(y + 1, x + 1);

            if (curVotes > threshold &&
                curVotes >  hist(y + 1, x) &&
                curVotes >= hist(y + 1, x + 2) &&
                curVotes >  hist(y, x + 1) &&
                curVotes >= hist(y + 2, x + 1))
            {
                const int ind = ::atomicAdd(&g_counter, 1);

                if (ind < maxSize)
                {
                    out[ind] = make_float4(x * dp, y * dp, 1.0f, 0.0f);
                    votes[ind] = make_int3(curVotes, 0, 0);
                }
            }
        }

        int GHT_Ballard_Pos_findPosInHist_gpu(PtrStepSzi hist, float4* out, int3* votes, int maxSize, float dp, int threshold)
        {
            void* counterPtr;
            cudaSafeCall( hipGetSymbolAddress(&counterPtr, HIP_SYMBOL(g_counter)) );

            cudaSafeCall( hipMemset(counterPtr, 0, sizeof(int)) );

            const dim3 block(32, 8);
            const dim3 grid(divUp(hist.cols - 2, block.x), divUp(hist.rows - 2, block.y));

            cudaSafeCall( hipFuncSetCacheConfig(reinterpret_cast<const void*>(GHT_Ballard_Pos_findPosInHist), hipFuncCachePreferL1) );

            GHT_Ballard_Pos_findPosInHist<<<grid, block>>>(hist, out, votes, maxSize, dp, threshold);
            cudaSafeCall( hipGetLastError() );

            cudaSafeCall( hipDeviceSynchronize() );

            int totalCount;
            cudaSafeCall( hipMemcpy(&totalCount, counterPtr, sizeof(int), hipMemcpyDeviceToHost) );

            totalCount = ::min(totalCount, maxSize);

            return totalCount;
        }

        ////////////////////////////////////////////////////////////////////////
        // GHT_Ballard_PosScale

        __global__ void GHT_Ballard_PosScale_calcHist(const unsigned int* coordList, const float* thetaList,
                                                      PtrStep<short2> r_table, const int* r_sizes,
                                                      PtrStepi hist, const int rows, const int cols,
                                                      const float minScale, const float scaleStep, const int scaleRange,
                                                      const float idp, const float thetaScale)
        {
            const unsigned int coord = coordList[blockIdx.x];
            float2 p;
            p.x = (coord & 0xFFFF);
            p.y = (coord >> 16) & 0xFFFF;

            const float theta = thetaList[blockIdx.x];
            const int n = __float2int_rn(theta * thetaScale);

            const short2* r_row = r_table.ptr(n);
            const int r_row_size = r_sizes[n];

            for (int j = 0; j < r_row_size; ++j)
            {
                const float2 d = saturate_cast<float2>(r_row[j]);

                for (int s = threadIdx.x; s < scaleRange; s += blockDim.x)
                {
                    const float scale = minScale + s * scaleStep;

                    float2 c = p - scale * d;

                    c.x *= idp;
                    c.y *= idp;

                    if (c.x >= 0 && c.x < cols && c.y >= 0 && c.y < rows)
                        ::atomicAdd(hist.ptr((s + 1) * (rows + 2) + __float2int_rn(c.y + 1)) + __float2int_rn(c.x + 1), 1);
                }
            }
        }

        void GHT_Ballard_PosScale_calcHist_gpu(const unsigned int* coordList, const float* thetaList, int pointsCount,
                                               PtrStepSz<short2> r_table, const int* r_sizes,
                                               PtrStepi hist, int rows, int cols,
                                               float minScale, float scaleStep, int scaleRange,
                                               float dp, int levels)
        {
            const dim3 block(256);
            const dim3 grid(pointsCount);

            const float idp = 1.0f / dp;
            const float thetaScale = levels / (2.0f * CV_PI_F);

            GHT_Ballard_PosScale_calcHist<<<grid, block>>>(coordList, thetaList,
                                                           r_table, r_sizes,
                                                           hist, rows, cols,
                                                           minScale, scaleStep, scaleRange,
                                                           idp, thetaScale);
            cudaSafeCall( hipGetLastError() );

            cudaSafeCall( hipDeviceSynchronize() );
        }

        __global__ void GHT_Ballard_PosScale_findPosInHist(const PtrStepi hist, const int rows, const int cols, const int scaleRange,
                                                           float4* out, int3* votes, const int maxSize,
                                                           const float minScale, const float scaleStep, const float dp, const int threshold)
        {
            const int x = blockIdx.x * blockDim.x + threadIdx.x;
            const int y = blockIdx.y * blockDim.y + threadIdx.y;

            if (x >= cols || y >= rows)
                return;

            for (int s = 0; s < scaleRange; ++s)
            {
                const float scale = minScale + s * scaleStep;

                const int prevScaleIdx = (s) * (rows + 2);
                const int curScaleIdx = (s + 1) * (rows + 2);
                const int nextScaleIdx = (s + 2) * (rows + 2);

                const int curVotes = hist(curScaleIdx + y + 1, x + 1);

                if (curVotes > threshold &&
                    curVotes >  hist(curScaleIdx + y + 1, x) &&
                    curVotes >= hist(curScaleIdx + y + 1, x + 2) &&
                    curVotes >  hist(curScaleIdx + y, x + 1) &&
                    curVotes >= hist(curScaleIdx + y + 2, x + 1) &&
                    curVotes >  hist(prevScaleIdx + y + 1, x + 1) &&
                    curVotes >= hist(nextScaleIdx + y + 1, x + 1))
                {
                    const int ind = ::atomicAdd(&g_counter, 1);

                    if (ind < maxSize)
                    {
                        out[ind] = make_float4(x * dp, y * dp, scale, 0.0f);
                        votes[ind] = make_int3(curVotes, curVotes, 0);
                    }
                }
            }
        }

        int GHT_Ballard_PosScale_findPosInHist_gpu(PtrStepi hist, int rows, int cols, int scaleRange, float4* out, int3* votes, int maxSize,
                                                   float minScale, float scaleStep, float dp, int threshold)
        {
            void* counterPtr;
            cudaSafeCall( hipGetSymbolAddress(&counterPtr, HIP_SYMBOL(g_counter)) );

            cudaSafeCall( hipMemset(counterPtr, 0, sizeof(int)) );

            const dim3 block(32, 8);
            const dim3 grid(divUp(cols, block.x), divUp(rows, block.y));

            cudaSafeCall( hipFuncSetCacheConfig(reinterpret_cast<const void*>(GHT_Ballard_PosScale_findPosInHist), hipFuncCachePreferL1) );

            GHT_Ballard_PosScale_findPosInHist<<<grid, block>>>(hist, rows, cols, scaleRange, out, votes, maxSize, minScale, scaleStep, dp, threshold);
            cudaSafeCall( hipGetLastError() );

            cudaSafeCall( hipDeviceSynchronize() );

            int totalCount;
            cudaSafeCall( hipMemcpy(&totalCount, counterPtr, sizeof(int), hipMemcpyDeviceToHost) );

            totalCount = ::min(totalCount, maxSize);

            return totalCount;
        }

        ////////////////////////////////////////////////////////////////////////
        // GHT_Ballard_PosRotation

        __global__ void GHT_Ballard_PosRotation_calcHist(const unsigned int* coordList, const float* thetaList,
                                                         PtrStep<short2> r_table, const int* r_sizes,
                                                         PtrStepi hist, const int rows, const int cols,
                                                         const float minAngle, const float angleStep, const int angleRange,
                                                         const float idp, const float thetaScale)
        {
            const unsigned int coord = coordList[blockIdx.x];
            float2 p;
            p.x = (coord & 0xFFFF);
            p.y = (coord >> 16) & 0xFFFF;

            const float thetaVal = thetaList[blockIdx.x];

            for (int a = threadIdx.x; a < angleRange; a += blockDim.x)
            {
                const float angle = (minAngle + a * angleStep) * (CV_PI_F / 180.0f);
                float sinA, cosA;
                sincosf(angle, &sinA, &cosA);

                float theta = thetaVal - angle;
                if (theta < 0)
                    theta += 2.0f * CV_PI_F;

                const int n = __float2int_rn(theta * thetaScale);

                const short2* r_row = r_table.ptr(n);
                const int r_row_size = r_sizes[n];

                for (int j = 0; j < r_row_size; ++j)
                {
                    const float2 d = saturate_cast<float2>(r_row[j]);

                    const float2 dr = make_float2(d.x * cosA - d.y * sinA, d.x * sinA + d.y * cosA);

                    float2 c = make_float2(p.x - dr.x, p.y - dr.y);
                    c.x *= idp;
                    c.y *= idp;

                    if (c.x >= 0 && c.x < cols && c.y >= 0 && c.y < rows)
                        ::atomicAdd(hist.ptr((a + 1) * (rows + 2) + __float2int_rn(c.y + 1)) + __float2int_rn(c.x + 1), 1);
                }
            }
        }

        void GHT_Ballard_PosRotation_calcHist_gpu(const unsigned int* coordList, const float* thetaList, int pointsCount,
                                                  PtrStepSz<short2> r_table, const int* r_sizes,
                                                  PtrStepi hist, int rows, int cols,
                                                  float minAngle, float angleStep, int angleRange,
                                                  float dp, int levels)
        {
            const dim3 block(256);
            const dim3 grid(pointsCount);

            const float idp = 1.0f / dp;
            const float thetaScale = levels / (2.0f * CV_PI_F);

            GHT_Ballard_PosRotation_calcHist<<<grid, block>>>(coordList, thetaList,
                                                              r_table, r_sizes,
                                                              hist, rows, cols,
                                                              minAngle, angleStep, angleRange,
                                                              idp, thetaScale);
            cudaSafeCall( hipGetLastError() );

            cudaSafeCall( hipDeviceSynchronize() );
        }

        __global__ void GHT_Ballard_PosRotation_findPosInHist(const PtrStepi hist, const int rows, const int cols, const int angleRange,
                                                              float4* out, int3* votes, const int maxSize,
                                                              const float minAngle, const float angleStep, const float dp, const int threshold)
        {
            const int x = blockIdx.x * blockDim.x + threadIdx.x;
            const int y = blockIdx.y * blockDim.y + threadIdx.y;

            if (x >= cols || y >= rows)
                return;

            for (int a = 0; a < angleRange; ++a)
            {
                const float angle = minAngle + a * angleStep;

                const int prevAngleIdx = (a) * (rows + 2);
                const int curAngleIdx = (a + 1) * (rows + 2);
                const int nextAngleIdx = (a + 2) * (rows + 2);

                const int curVotes = hist(curAngleIdx + y + 1, x + 1);

                if (curVotes > threshold &&
                    curVotes >  hist(curAngleIdx + y + 1, x) &&
                    curVotes >= hist(curAngleIdx + y + 1, x + 2) &&
                    curVotes >  hist(curAngleIdx + y, x + 1) &&
                    curVotes >= hist(curAngleIdx + y + 2, x + 1) &&
                    curVotes >  hist(prevAngleIdx + y + 1, x + 1) &&
                    curVotes >= hist(nextAngleIdx + y + 1, x + 1))
                {
                    const int ind = ::atomicAdd(&g_counter, 1);

                    if (ind < maxSize)
                    {
                        out[ind] = make_float4(x * dp, y * dp, 1.0f, angle);
                        votes[ind] = make_int3(curVotes, 0, curVotes);
                    }
                }
            }
        }

        int GHT_Ballard_PosRotation_findPosInHist_gpu(PtrStepi hist, int rows, int cols, int angleRange, float4* out, int3* votes, int maxSize,
                                                      float minAngle, float angleStep, float dp, int threshold)
        {
            void* counterPtr;
            cudaSafeCall( hipGetSymbolAddress(&counterPtr, HIP_SYMBOL(g_counter)) );

            cudaSafeCall( hipMemset(counterPtr, 0, sizeof(int)) );

            const dim3 block(32, 8);
            const dim3 grid(divUp(cols, block.x), divUp(rows, block.y));

            cudaSafeCall( hipFuncSetCacheConfig(reinterpret_cast<const void*>(GHT_Ballard_PosRotation_findPosInHist), hipFuncCachePreferL1) );

            GHT_Ballard_PosRotation_findPosInHist<<<grid, block>>>(hist, rows, cols, angleRange, out, votes, maxSize, minAngle, angleStep, dp, threshold);
            cudaSafeCall( hipGetLastError() );

            cudaSafeCall( hipDeviceSynchronize() );

            int totalCount;
            cudaSafeCall( hipMemcpy(&totalCount, counterPtr, sizeof(int), hipMemcpyDeviceToHost) );

            totalCount = ::min(totalCount, maxSize);

            return totalCount;
        }

        ////////////////////////////////////////////////////////////////////////
        // GHT_Guil_Full

        struct FeatureTable
        {
            uchar* p1_pos_data;
            size_t p1_pos_step;

            uchar* p1_theta_data;
            size_t p1_theta_step;

            uchar* p2_pos_data;
            size_t p2_pos_step;

            uchar* d12_data;
            size_t d12_step;

            uchar* r1_data;
            size_t r1_step;

            uchar* r2_data;
            size_t r2_step;
        };

        __constant__ FeatureTable c_templFeatures;
        __constant__ FeatureTable c_imageFeatures;

        void GHT_Guil_Full_setTemplFeatures(PtrStepb p1_pos, PtrStepb p1_theta, PtrStepb p2_pos, PtrStepb d12, PtrStepb r1, PtrStepb r2)
        {
            FeatureTable tbl;

            tbl.p1_pos_data = p1_pos.data;
            tbl.p1_pos_step = p1_pos.step;

            tbl.p1_theta_data = p1_theta.data;
            tbl.p1_theta_step = p1_theta.step;

            tbl.p2_pos_data = p2_pos.data;
            tbl.p2_pos_step = p2_pos.step;

            tbl.d12_data = d12.data;
            tbl.d12_step = d12.step;

            tbl.r1_data = r1.data;
            tbl.r1_step = r1.step;

            tbl.r2_data = r2.data;
            tbl.r2_step = r2.step;

            cudaSafeCall( hipMemcpyToSymbol(HIP_SYMBOL(c_templFeatures), &tbl, sizeof(FeatureTable)) );
        }
        void GHT_Guil_Full_setImageFeatures(PtrStepb p1_pos, PtrStepb p1_theta, PtrStepb p2_pos, PtrStepb d12, PtrStepb r1, PtrStepb r2)
        {
            FeatureTable tbl;

            tbl.p1_pos_data = p1_pos.data;
            tbl.p1_pos_step = p1_pos.step;

            tbl.p1_theta_data = p1_theta.data;
            tbl.p1_theta_step = p1_theta.step;

            tbl.p2_pos_data = p2_pos.data;
            tbl.p2_pos_step = p2_pos.step;

            tbl.d12_data = d12.data;
            tbl.d12_step = d12.step;

            tbl.r1_data = r1.data;
            tbl.r1_step = r1.step;

            tbl.r2_data = r2.data;
            tbl.r2_step = r2.step;

            cudaSafeCall( hipMemcpyToSymbol(HIP_SYMBOL(c_imageFeatures), &tbl, sizeof(FeatureTable)) );
        }

        struct TemplFeatureTable
        {
            static __device__ float2* p1_pos(int n)
            {
                return (float2*)(c_templFeatures.p1_pos_data + n * c_templFeatures.p1_pos_step);
            }
            static __device__ float* p1_theta(int n)
            {
                return (float*)(c_templFeatures.p1_theta_data + n * c_templFeatures.p1_theta_step);
            }
            static __device__ float2* p2_pos(int n)
            {
                return (float2*)(c_templFeatures.p2_pos_data + n * c_templFeatures.p2_pos_step);
            }

            static __device__ float* d12(int n)
            {
                return (float*)(c_templFeatures.d12_data + n * c_templFeatures.d12_step);
            }

            static __device__ float2* r1(int n)
            {
                return (float2*)(c_templFeatures.r1_data + n * c_templFeatures.r1_step);
            }
            static __device__ float2* r2(int n)
            {
                return (float2*)(c_templFeatures.r2_data + n * c_templFeatures.r2_step);
            }
        };
        struct ImageFeatureTable
        {
            static __device__ float2* p1_pos(int n)
            {
                return (float2*)(c_imageFeatures.p1_pos_data + n * c_imageFeatures.p1_pos_step);
            }
            static __device__ float* p1_theta(int n)
            {
                return (float*)(c_imageFeatures.p1_theta_data + n * c_imageFeatures.p1_theta_step);
            }
            static __device__ float2* p2_pos(int n)
            {
                return (float2*)(c_imageFeatures.p2_pos_data + n * c_imageFeatures.p2_pos_step);
            }

            static __device__ float* d12(int n)
            {
                return (float*)(c_imageFeatures.d12_data + n * c_imageFeatures.d12_step);
            }

            static __device__ float2* r1(int n)
            {
                return (float2*)(c_imageFeatures.r1_data + n * c_imageFeatures.r1_step);
            }
            static __device__ float2* r2(int n)
            {
                return (float2*)(c_imageFeatures.r2_data + n * c_imageFeatures.r2_step);
            }
        };

        __device__ float clampAngle(float a)
        {
            float res = a;

            while (res > 2.0f * CV_PI_F)
                res -= 2.0f * CV_PI_F;
            while (res < 0.0f)
                res += 2.0f * CV_PI_F;

            return res;
        }

        __device__ bool angleEq(float a, float b, float eps)
        {
            return (::fabs(clampAngle(a - b)) <= eps);
        }

        template <class FT, bool isTempl>
        __global__ void GHT_Guil_Full_buildFeatureList(const unsigned int* coordList, const float* thetaList, const int pointsCount,
                                                       int* sizes, const int maxSize,
                                                       const float xi, const float angleEpsilon, const float alphaScale,
                                                       const float2 center, const float maxDist)
        {
            const float p1_theta = thetaList[blockIdx.x];
            const unsigned int coord1 = coordList[blockIdx.x];
            float2 p1_pos;
            p1_pos.x = (coord1 & 0xFFFF);
            p1_pos.y = (coord1 >> 16) & 0xFFFF;

            for (int i = threadIdx.x; i < pointsCount; i += blockDim.x)
            {
                const float p2_theta = thetaList[i];
                const unsigned int coord2 = coordList[i];
                float2 p2_pos;
                p2_pos.x = (coord2 & 0xFFFF);
                p2_pos.y = (coord2 >> 16) & 0xFFFF;

                if (angleEq(p1_theta - p2_theta, xi, angleEpsilon))
                {
                    const float2 d = p1_pos - p2_pos;

                    float alpha12 = clampAngle(::atan2(d.y, d.x) - p1_theta);
                    float d12 = ::sqrtf(d.x * d.x + d.y * d.y);

                    if (d12 > maxDist)
                        continue;

                    float2 r1 = p1_pos - center;
                    float2 r2 = p2_pos - center;

                    const int n = __float2int_rn(alpha12 * alphaScale);

                    const int ind = ::atomicAdd(sizes + n, 1);

                    if (ind < maxSize)
                    {
                        if (!isTempl)
                        {
                            FT::p1_pos(n)[ind] = p1_pos;
                            FT::p2_pos(n)[ind] = p2_pos;
                        }

                        FT::p1_theta(n)[ind] = p1_theta;

                        FT::d12(n)[ind] = d12;

                        if (isTempl)
                        {
                            FT::r1(n)[ind] = r1;
                            FT::r2(n)[ind] = r2;
                        }
                    }
                }
            }
        }

        template <class FT, bool isTempl>
        void GHT_Guil_Full_buildFeatureList_caller(const unsigned int* coordList, const float* thetaList, int pointsCount,
                                                   int* sizes, int maxSize,
                                                   float xi, float angleEpsilon, int levels,
                                                   float2 center, float maxDist)
        {
            const dim3 block(256);
            const dim3 grid(pointsCount);

            const float alphaScale = levels / (2.0f * CV_PI_F);

            GHT_Guil_Full_buildFeatureList<FT, isTempl><<<grid, block>>>(coordList, thetaList, pointsCount,
                                                                         sizes, maxSize,
                                                                         xi * (CV_PI_F / 180.0f), angleEpsilon * (CV_PI_F / 180.0f), alphaScale,
                                                                         center, maxDist);
            cudaSafeCall( hipGetLastError() );

            cudaSafeCall( hipDeviceSynchronize() );

            thrust::device_ptr<int> sizesPtr(sizes);
            thrust::transform(sizesPtr, sizesPtr + levels + 1, sizesPtr, device::bind2nd(device::minimum<int>(), maxSize));
        }

        void GHT_Guil_Full_buildTemplFeatureList_gpu(const unsigned int* coordList, const float* thetaList, int pointsCount,
                                                     int* sizes, int maxSize,
                                                     float xi, float angleEpsilon, int levels,
                                                     float2 center, float maxDist)
        {
            GHT_Guil_Full_buildFeatureList_caller<TemplFeatureTable, true>(coordList, thetaList, pointsCount,
                                                                           sizes, maxSize,
                                                                           xi, angleEpsilon, levels,
                                                                           center, maxDist);
        }
        void GHT_Guil_Full_buildImageFeatureList_gpu(const unsigned int* coordList, const float* thetaList, int pointsCount,
                                                     int* sizes, int maxSize,
                                                     float xi, float angleEpsilon, int levels,
                                                     float2 center, float maxDist)
        {
            GHT_Guil_Full_buildFeatureList_caller<ImageFeatureTable, false>(coordList, thetaList, pointsCount,
                                                                            sizes, maxSize,
                                                                            xi, angleEpsilon, levels,
                                                                            center, maxDist);
        }

        __global__ void GHT_Guil_Full_calcOHist(const int* templSizes, const int* imageSizes, int* OHist,
                                                const float minAngle, const float maxAngle, const float iAngleStep, const int angleRange)
        {
            extern __shared__ int s_OHist[];
            for (int i = threadIdx.x; i <= angleRange; i += blockDim.x)
                s_OHist[i] = 0;
            __syncthreads();

            const int tIdx = blockIdx.x;
            const int level = blockIdx.y;

            const int tSize = templSizes[level];

            if (tIdx < tSize)
            {
                const int imSize = imageSizes[level];

                const float t_p1_theta = TemplFeatureTable::p1_theta(level)[tIdx];

                for (int i = threadIdx.x; i < imSize; i += blockDim.x)
                {
                    const float im_p1_theta = ImageFeatureTable::p1_theta(level)[i];

                    const float angle = clampAngle(im_p1_theta - t_p1_theta);

                    if (angle >= minAngle && angle <= maxAngle)
                    {
                        const int n = __float2int_rn((angle - minAngle) * iAngleStep);
                        Emulation::smem::atomicAdd(&s_OHist[n], 1);
                    }
                }
            }
            __syncthreads();

            for (int i = threadIdx.x; i <= angleRange; i += blockDim.x)
                ::atomicAdd(OHist + i, s_OHist[i]);
        }

        void GHT_Guil_Full_calcOHist_gpu(const int* templSizes, const int* imageSizes, int* OHist,
                                         float minAngle, float maxAngle, float angleStep, int angleRange,
                                         int levels, int tMaxSize)
        {
            const dim3 block(256);
            const dim3 grid(tMaxSize, levels + 1);

            minAngle *= (CV_PI_F / 180.0f);
            maxAngle *= (CV_PI_F / 180.0f);
            angleStep *= (CV_PI_F / 180.0f);

            const size_t smemSize = (angleRange + 1) * sizeof(float);

            GHT_Guil_Full_calcOHist<<<grid, block, smemSize>>>(templSizes, imageSizes, OHist,
                                                               minAngle, maxAngle, 1.0f / angleStep, angleRange);
            cudaSafeCall( hipGetLastError() );

            cudaSafeCall( hipDeviceSynchronize() );
        }

        __global__ void GHT_Guil_Full_calcSHist(const int* templSizes, const int* imageSizes, int* SHist,
                                                const float angle, const float angleEpsilon,
                                                const float minScale, const float maxScale, const float iScaleStep, const int scaleRange)
        {
            extern __shared__ int s_SHist[];
            for (int i = threadIdx.x; i <= scaleRange; i += blockDim.x)
                s_SHist[i] = 0;
            __syncthreads();

            const int tIdx = blockIdx.x;
            const int level = blockIdx.y;

            const int tSize = templSizes[level];

            if (tIdx < tSize)
            {
                const int imSize = imageSizes[level];

                const float t_p1_theta = TemplFeatureTable::p1_theta(level)[tIdx] + angle;
                const float t_d12 = TemplFeatureTable::d12(level)[tIdx] + angle;

                for (int i = threadIdx.x; i < imSize; i += blockDim.x)
                {
                    const float im_p1_theta = ImageFeatureTable::p1_theta(level)[i];
                    const float im_d12 = ImageFeatureTable::d12(level)[i];

                    if (angleEq(im_p1_theta, t_p1_theta, angleEpsilon))
                    {
                        const float scale = im_d12 / t_d12;

                        if (scale >= minScale && scale <= maxScale)
                        {
                            const int s = __float2int_rn((scale - minScale) * iScaleStep);
                            Emulation::smem::atomicAdd(&s_SHist[s], 1);
                        }
                    }
                }
            }
            __syncthreads();

            for (int i = threadIdx.x; i <= scaleRange; i += blockDim.x)
                ::atomicAdd(SHist + i, s_SHist[i]);
        }

        void GHT_Guil_Full_calcSHist_gpu(const int* templSizes, const int* imageSizes, int* SHist,
                                         float angle, float angleEpsilon,
                                         float minScale, float maxScale, float iScaleStep, int scaleRange,
                                         int levels, int tMaxSize)
        {
            const dim3 block(256);
            const dim3 grid(tMaxSize, levels + 1);

            angle *= (CV_PI_F / 180.0f);
            angleEpsilon *= (CV_PI_F / 180.0f);

            const size_t smemSize = (scaleRange + 1) * sizeof(float);

            GHT_Guil_Full_calcSHist<<<grid, block, smemSize>>>(templSizes, imageSizes, SHist,
                                                               angle, angleEpsilon,
                                                               minScale, maxScale, iScaleStep, scaleRange);
            cudaSafeCall( hipGetLastError() );

            cudaSafeCall( hipDeviceSynchronize() );
        }

        __global__ void GHT_Guil_Full_calcPHist(const int* templSizes, const int* imageSizes, PtrStepSzi PHist,
                                                const float angle, const float sinVal, const float cosVal, const float angleEpsilon, const float scale,
                                                const float idp)
        {
            const int tIdx = blockIdx.x;
            const int level = blockIdx.y;

            const int tSize = templSizes[level];

            if (tIdx < tSize)
            {
                const int imSize = imageSizes[level];

                const float t_p1_theta = TemplFeatureTable::p1_theta(level)[tIdx] + angle;

                float2 r1 = TemplFeatureTable::r1(level)[tIdx];
                float2 r2 = TemplFeatureTable::r2(level)[tIdx];

                r1 = r1 * scale;
                r2 = r2 * scale;

                r1 = make_float2(cosVal * r1.x - sinVal * r1.y, sinVal * r1.x + cosVal * r1.y);
                r2 = make_float2(cosVal * r2.x - sinVal * r2.y, sinVal * r2.x + cosVal * r2.y);

                for (int i = threadIdx.x; i < imSize; i += blockDim.x)
                {
                    const float im_p1_theta = ImageFeatureTable::p1_theta(level)[i];

                    const float2 im_p1_pos = ImageFeatureTable::p1_pos(level)[i];
                    const float2 im_p2_pos = ImageFeatureTable::p2_pos(level)[i];

                    if (angleEq(im_p1_theta, t_p1_theta, angleEpsilon))
                    {
                        float2 c1, c2;

                        c1 = im_p1_pos - r1;
                        c1 = c1 * idp;

                        c2 = im_p2_pos - r2;
                        c2 = c2 * idp;

                        if (::fabs(c1.x - c2.x) > 1 || ::fabs(c1.y - c2.y) > 1)
                            continue;

                        if (c1.y >= 0 && c1.y < PHist.rows - 2 && c1.x >= 0 && c1.x < PHist.cols - 2)
                            ::atomicAdd(PHist.ptr(__float2int_rn(c1.y) + 1) + __float2int_rn(c1.x) + 1, 1);
                    }
                }
            }
        }

        void GHT_Guil_Full_calcPHist_gpu(const int* templSizes, const int* imageSizes, PtrStepSzi PHist,
                                         float angle, float angleEpsilon, float scale,
                                         float dp,
                                         int levels, int tMaxSize)
        {
            const dim3 block(256);
            const dim3 grid(tMaxSize, levels + 1);

            angle *= (CV_PI_F / 180.0f);
            angleEpsilon *= (CV_PI_F / 180.0f);

            const float sinVal = ::sinf(angle);
            const float cosVal = ::cosf(angle);

            cudaSafeCall( hipFuncSetCacheConfig(reinterpret_cast<const void*>(GHT_Guil_Full_calcPHist), hipFuncCachePreferL1) );

            GHT_Guil_Full_calcPHist<<<grid, block>>>(templSizes, imageSizes, PHist,
                                                     angle, sinVal, cosVal, angleEpsilon, scale,
                                                     1.0f / dp);
            cudaSafeCall( hipGetLastError() );

            cudaSafeCall( hipDeviceSynchronize() );
        }

        __global__ void GHT_Guil_Full_findPosInHist(const PtrStepSzi hist, float4* out, int3* votes, const int maxSize,
                                                    const float angle, const int angleVotes, const float scale, const int scaleVotes,
                                                    const float dp, const int threshold)
        {
            const int x = blockIdx.x * blockDim.x + threadIdx.x;
            const int y = blockIdx.y * blockDim.y + threadIdx.y;

            if (x >= hist.cols - 2 || y >= hist.rows - 2)
                return;

            const int curVotes = hist(y + 1, x + 1);

            if (curVotes > threshold &&
                curVotes >  hist(y + 1, x) &&
                curVotes >= hist(y + 1, x + 2) &&
                curVotes >  hist(y, x + 1) &&
                curVotes >= hist(y + 2, x + 1))
            {
                const int ind = ::atomicAdd(&g_counter, 1);

                if (ind < maxSize)
                {
                    out[ind] = make_float4(x * dp, y * dp, scale, angle);
                    votes[ind] = make_int3(curVotes, scaleVotes, angleVotes);
                }
            }
        }

        int GHT_Guil_Full_findPosInHist_gpu(PtrStepSzi hist, float4* out, int3* votes, int curSize, int maxSize,
                                             float angle, int angleVotes, float scale, int scaleVotes,
                                             float dp, int threshold)
        {
            void* counterPtr;
            cudaSafeCall( hipGetSymbolAddress(&counterPtr, HIP_SYMBOL(g_counter)) );

            cudaSafeCall( hipMemcpy(counterPtr, &curSize, sizeof(int), hipMemcpyHostToDevice) );

            const dim3 block(32, 8);
            const dim3 grid(divUp(hist.cols - 2, block.x), divUp(hist.rows - 2, block.y));

            cudaSafeCall( hipFuncSetCacheConfig(reinterpret_cast<const void*>(GHT_Guil_Full_findPosInHist), hipFuncCachePreferL1) );

            GHT_Guil_Full_findPosInHist<<<grid, block>>>(hist, out, votes, maxSize,
                                                         angle, angleVotes, scale, scaleVotes,
                                                         dp, threshold);
            cudaSafeCall( hipGetLastError() );

            cudaSafeCall( hipDeviceSynchronize() );

            int totalCount;
            cudaSafeCall( hipMemcpy(&totalCount, counterPtr, sizeof(int), hipMemcpyDeviceToHost) );

            totalCount = ::min(totalCount, maxSize);

            return totalCount;
        }
    }
}}}


#endif /* CUDA_DISABLER */
